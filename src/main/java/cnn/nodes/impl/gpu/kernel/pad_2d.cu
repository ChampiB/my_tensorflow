
#include <hip/hip_runtime.h>
/**
 * Compute the padding activation.
 * @param x_shape the shape of the input buffer.
 * @param x the input buffer.
 * @param y_shape the shape of the output buffer.
 * @param y the output buffer.
 * @param value the padding value.
 * @return nothing.
 */
extern "C"
__global__ void activation(long *x_shape, float *x, long *y_shape, float *y, float value)
{
    int yfs = y_shape[2] * y_shape[3]; // Y feature size.
    int y_index = threadIdx.x * y_shape[1] * yfs + blockIdx.x * yfs + blockIdx.y * y_shape[3] + blockIdx.z;
    if (threadIdx.x >= x_shape[0] || blockIdx.x >= x_shape[1] || blockIdx.y >= x_shape[2] || blockIdx.z >= x_shape[3]) {
        y[y_index] = value;
    } else {
        int xfs = x_shape[2] * x_shape[3]; // X feature size.
        int x_index = threadIdx.x * x_shape[1] * xfs + blockIdx.x * xfs + blockIdx.y * x_shape[3] + blockIdx.z;
        y[y_index] = x[x_index];
    }
}
// (int)ycShape[1], (int)ycShape[2], (int)ycShape[3],
// (int)ycShape[0], 1, 1,
/**
 * Compute the gradients with respect to the inputs.
 * @param x_shape the shape of the input buffer.
 * @param x the input buffer.
 * @param y_shape the shape of the output buffer.
 * @param y the output buffer.
 * @return nothing.
 */
extern "C"
__global__ void inputs_gradients(long *x_shape, float *x, long *y_shape, float *y)
{
    if (threadIdx.x < y_shape[0] && blockIdx.x < y_shape[1] && blockIdx.y < y_shape[2] && blockIdx.z < y_shape[3]) {
        int yfs = y_shape[2] * y_shape[3]; // Y feature size.
        int y_index = threadIdx.x * y_shape[1] * yfs + blockIdx.x * yfs + blockIdx.y * y_shape[3] + blockIdx.y;
        int xfs = x_shape[2] * x_shape[3]; // X feature size.
        int x_index = threadIdx.x * x_shape[1] * xfs + blockIdx.x * xfs + blockIdx.y * x_shape[3] + blockIdx.y;
        y[y_index] = x[x_index];
    }
}

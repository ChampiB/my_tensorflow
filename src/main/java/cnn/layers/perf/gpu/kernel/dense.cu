
#include <hip/hip_runtime.h>
/**
 * Configuration indexes.
 */
#define NUMBER_OF_IMAGES conf[0]
#define NUMBER_OF_INPUTS conf[1]
#define NUMBER_OF_OUTPUTS conf[2]

/**
 * The memory shared between the threads of each block.
 */
extern __shared__ float sdata[];

/**
 * Compute the sum of the array's elements.
 * @param sdata the array.
 * @return the sum.
 */
__device__ float reduce_sum(float *sdata)
{
    for (int i = 1; i < blockDim.x; i++) {
        sdata[0] += sdata[i];
    }
    return sdata[0];
}

/**
 * Compute the convolution activation.
 * @param conf is the configuration of the kernel.
 * @param x is the input activation.
 * @param w is the weights of the layer.
 * @param y is the output of the layer.
 * @return nothing.
 */
extern "C"
__global__ void activation(int *conf, float *x, float *w, float *y)
{
    int bid = blockIdx.x * NUMBER_OF_OUTPUTS + blockIdx.y;
    sdata[threadIdx.x] = 0;
    for (int i = threadIdx.x; i < NUMBER_OF_INPUTS; i += blockDim.x) {
        sdata[threadIdx.x] += x[blockIdx.x * NUMBER_OF_INPUTS + i] * w[i * gridDim.y + blockIdx.y];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        y[bid] = reduce_sum(sdata);
    }
}

/**
 * Compute the gradients with respect to the weights.
 * @param conf is the configuration of the kernel.
 * @param x is the input activation.
 * @param g is the gradients with respect to the output.
 * @param r is the weights gradients, i.e. output buffer.
 * @return nothing.
 */
extern "C"
__global__ void weights_gradients(int *conf, float *x, float *g, float *r)
{
    int tid = threadIdx.x;
    sdata[tid] = 0;
    for (int ii = threadIdx.x; ii < NUMBER_OF_IMAGES; ii += blockDim.x) {
        sdata[tid] += x[ii * NUMBER_OF_INPUTS + blockIdx.y] * g[ii * NUMBER_OF_OUTPUTS + blockIdx.x];
    }
    __syncthreads();

    if (tid == 0) {
        r[blockIdx.y * NUMBER_OF_OUTPUTS + blockIdx.x] = reduce_sum(sdata);
    }
}

/**
 * Compute the gradients with respect to the weights.
 * @param conf is the configuration of the kernel.
 * @param w is the weights of the layer.
 * @param g is the gradients with respect to the output.
 * @param r is the weights gradients, i.e. output buffer.
 * @return nothing.
 */
extern "C"
__global__ void inputs_gradients(int *conf, float *w, float *g, float *r)
{
    sdata[threadIdx.x] = 0;
    for (int i = threadIdx.x; i < NUMBER_OF_IMAGES; i += blockDim.x) {
        sdata[threadIdx.x] +=
            w[i * NUMBER_OF_INPUTS + blockIdx.y + 1] * g[i * NUMBER_OF_OUTPUTS + blockIdx.y + 1];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        r[blockIdx.y * gridDim.x + blockIdx.x] = reduce_sum(sdata);
    }
}
